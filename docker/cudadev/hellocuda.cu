
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>

void hello_from_cpu(void) { printf("Hello from your CPU\n"); }

__global__ void hello_from_gpu(void) { printf("Hello from the GPU\n"); }

// Host Code
int main(void) {

  hello_from_cpu();
  hello_from_gpu<<<1, 10>>>();
  hipDeviceReset();
  return EXIT_SUCCESS;
}
